#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required point_b_y applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <iomanip>
#include <memory>
#include <ostream>
#include <type_traits>
#include <cudf/types.hpp>
#include <cudf/utilities/type_dispatcher.hpp>
#include <cudf/column/column.hpp>
#include <cudf/column/column_device_view.cuh>
#include <cudf/column/column_factories.hpp>
#include <cudf/column/column_view.hpp>
#include <cuspatial/error.hpp>
#include <cudf/utilities/error.hpp>
#include <rmm/mr/device/device_memory_resource.hpp>
#include <rmm/thrust_rmm_allocator.h>
#include <rmm/device_buffer.hpp>

#include <thrust/iterator/transform_output_iterator.h>
#include <thrust/iterator/discard_iterator.h>
#include <thrust/gather.h>
#include <limits>
#include <iterator>


template <class OutputIterator>
class haus_output_iterator_proxy;

// namespace thrust {
// namespace detail {
//     template <class OutputIterator>
//     struct is_proxy_reference<haus_output_iterator_proxy<OutputIterator>>
//         : public thrust::detail::true_type {};
// }
// }

namespace {

using size_type = int32_t;

template<typename T>
constexpr auto magnitude_squared(T a, T b) {
    return a * a + b * b;
}

template<typename T>
std::unique_ptr<cudf::column> make_column(
    size_type size,
    hipStream_t stream = 0,
    rmm::mr::device_memory_resource* mr = rmm::mr::get_default_resource()
)
{
    auto tid = cudf::type_to_id<T>();

    return cudf::make_fixed_width_column(
        cudf::data_type{ tid },
        size,
        cudf::mask_state::UNALLOCATED,
        stream,
        mr);
}

template<typename T>
using haus = thrust::tuple<int32_t, int32_t, int32_t, int32_t, T, T, T, int64_t>;

template<typename T> __device__ int32_t haus_col(haus<T> value) { return thrust::get<0>(value); }
template<typename T> __device__ int32_t haus_row(haus<T> value) { return thrust::get<1>(value); }
template<typename T> __device__ int32_t cell_min(haus<T> value) { return thrust::get<2>(value); }
template<typename T> __device__ int32_t cell_max(haus<T> value) { return thrust::get<3>(value); }
template<typename T> __device__ T haus_max(haus<T> value) { return thrust::get<4>(value); }
template<typename T> __device__ T haus_min(haus<T> value) { return thrust::get<5>(value); }
template<typename T> __device__ T haus_res(haus<T> value) { return thrust::get<6>(value); }
template<typename T> __device__ int64_t haus_dst(haus<T> value) { return thrust::get<7>(value); }

template<typename T>
struct haus_key_compare
{
    bool __device__ operator()(haus<T> a, haus<T> b)
    {
        return haus_col(a) == haus_col(b)
            && haus_row(a) == haus_row(b);
    }
};

template<typename T>
struct haus_reduce
{
    haus<T> __device__ operator()(haus<T> lhs, haus<T> rhs)
    {
        T new_min{};
        T new_max{};

        if (cell_max(lhs) == cell_min(rhs))
        {
            new_min = min(haus_min(lhs), haus_min(rhs));
            new_max = haus_max(lhs);
        }
        else
        {
            new_min = haus_min(rhs);
            new_max = max(haus_res(lhs), haus_max(rhs));
        }

        return haus<T>{
            haus_col(lhs),
            haus_row(lhs),
            cell_min(lhs),
            cell_max(rhs),
            new_max,
            new_min,
            max(new_max, new_min),
            haus_dst(rhs)
        };
    }
};

struct size_from_offsets_functor
{
    cudf::column_device_view offsets;
    size_type end;

    size_type __device__ operator()(size_type idx)
    {
        auto curr_offset = offsets.element<size_type>(idx);
        auto next_idx = idx + 1;
        auto next_offset = next_idx >= offsets.size()
            ? end
            : offsets.element<size_type>(next_idx);

        return next_offset - curr_offset;
    }
};

template <typename OutputIterator>
class haus_output_iterator_proxy
{
  public:
    __host__ __device__
    haus_output_iterator_proxy(const OutputIterator& out, const OutputIterator& begin) : out(out), begin(begin)
    {
    }

    __thrust_exec_check_disable__
    template <typename T>
    __host__ __device__
    haus_output_iterator_proxy operator=(const T& x)
    {
        if (haus_dst(x) >= 0) {
            *(begin + haus_dst(x)) = x;
        }

        return *this;
    }

  private:
    OutputIterator out;
    OutputIterator begin;
};

template<typename OutputIterator>
class haus_output_iterator;

template <typename OutputIterator>
struct haus_output_iterator_base
{
    typedef thrust::iterator_adaptor
    <
        haus_output_iterator<OutputIterator>
      , OutputIterator
      , thrust::use_default
      , thrust::use_default
      , thrust::use_default
      , haus_output_iterator_proxy<OutputIterator>
    > type;
};

template<typename OutputIterator>
class haus_output_iterator : public haus_output_iterator_base<OutputIterator>::type
{
public:
    typedef typename
    haus_output_iterator_base<OutputIterator>::type
    super_t;

    friend class thrust::iterator_core_access;

    __host__ __device__
    haus_output_iterator(OutputIterator const& out, OutputIterator const& begin) : super_t(out), begin(begin) {}
    
 private:
    __host__ __device__
    typename super_t::reference dereference() const
    {
        return haus_output_iterator_proxy<OutputIterator>(this->base_reference(), begin);
    }

    OutputIterator begin;
};

template <typename OutputIterator>
haus_output_iterator<OutputIterator>
__host__ __device__
make_haus_output_iterator(OutputIterator out)
{
    return haus_output_iterator<OutputIterator>(out, out);
}

template<typename T, typename SpaceSizeIterator>
struct haus_travesal
{
    int64_t num_spaces;
    int64_t n;
    size_type const* o;
    size_type const* l;
    SpaceSizeIterator const s;
    cudf::column_device_view xs;
    cudf::column_device_view ys;

    haus<T> __device__ operator()(int64_t idx)
    {
        // ===== Reduction Key ===========
        int64_t haus_col = l[idx / n];
        int64_t ox = o[haus_col];
        int64_t sx = s[haus_col];
        int64_t ox_n = ox * n;

        int64_t haus_row = l[(idx - ox_n) / sx];
        int64_t oy = o[haus_row];
        int64_t sy = s[haus_row];

        // ===== Min/Max Key ==========
        int64_t haus_offset = ox_n + sx * oy;
        int64_t cell_idx = idx - haus_offset;
        int64_t cell_col = cell_idx / sy;

        // ===== Distance =============
        int64_t cell_offset = ox_n + oy + (n - sy) * cell_col + cell_idx;
        int64_t col = cell_offset / n;
        int64_t row = cell_offset % n;
        T a_x = xs.element<T>(row);
        T a_y = ys.element<T>(row);
        T b_x = xs.element<T>(col);
        T b_y = ys.element<T>(col);

        double distance_d = hypot(static_cast<double>(b_x - a_x),
                                  static_cast<double>(b_y - a_y));

        T distance = static_cast<T>(distance_d);

        // int64_t distance = abs(b_x - a_x);

        // int64_t elm = ox_n + sx * oy + sx * sy - 1;
        // int64_t dst = haus_col * num_spaces + haus_row;
        int64_t elm = ox_n + (sx - 1) * n + oy + sy - 1;
        // ===== All ==================
        return haus<T>{
            haus_col,
            haus_row,
            cell_col,
            cell_col,
            0,
            distance,
            distance,
            elm == cell_offset ? haus_col * num_spaces + haus_row : -1
        };
    }
};

struct hausdorff_functor
{
    template<typename T, typename... Args>
    std::enable_if_t<not std::is_floating_point<T>::value, std::unique_ptr<cudf::column>>
    operator()(Args&&...)
    {
        CUSPATIAL_FAIL("Non-floating point operation is not supported");
    }

    template<typename T>
    std::enable_if_t<std::is_floating_point<T>::value, std::unique_ptr<cudf::column>>
    operator()(cudf::column_view const& xs,
               cudf::column_view const& ys,
               cudf::column_view const& space_offsets,
               rmm::mr::device_memory_resource *mr,
               hipStream_t stream)
    {
        size_type num_points = xs.size();
        size_type num_spaces = space_offsets.size();
        int64_t num_results = static_cast<int64_t>(num_spaces) * static_cast<int64_t>(num_spaces);

        if (num_results == 0)
        {
            return make_column<T>(0, stream, mr);
        }

        // ===== Make Space Lookup ================================================================

        auto temp_space_lookup = rmm::device_vector<size_type>(num_points);

        thrust::scatter(
            rmm::exec_policy(stream)->on(stream),
            thrust::make_constant_iterator(1),
            thrust::make_constant_iterator(1) + num_spaces - 1,
            space_offsets.begin<size_type>() + 1,
            temp_space_lookup.begin()
        );

        thrust::inclusive_scan(
            rmm::exec_policy(stream)->on(stream),
            temp_space_lookup.cbegin(),
            temp_space_lookup.cend(),
            temp_space_lookup.begin()
        );

        // ===== Make Space Size Iterator =========================================================

        auto count = thrust::make_counting_iterator<int64_t>(0);

        auto d_space_offsets = cudf::column_device_view::create(space_offsets);

        auto space_offset_iterator = thrust::make_transform_iterator(
            count,
            size_from_offsets_functor { *d_space_offsets, xs.size() }
        );

        // ===== Make Cartesian Distances =========================================================

        auto d_xs = cudf::column_device_view::create(xs);
        auto d_ys = cudf::column_device_view::create(ys);

        auto num_cartesian = static_cast<int64_t>(num_points) * static_cast<int64_t>(num_points);

        auto hausdorff_iter = thrust::make_transform_iterator(
            count,
            haus_travesal<T, decltype(space_offset_iterator)>{
                num_spaces,
                num_points,
                space_offsets.data<size_type>(),
                temp_space_lookup.data().get(),
                space_offset_iterator,
                *d_xs,
                *d_ys
            }
        );

        // ===== Calculate ========================================================================

        std::unique_ptr<cudf::column> result = make_column<T>(num_results, stream, mr);

        auto out_real = result->mutable_view().begin<T>();

        auto discard_buffer = rmm::device_buffer(sizeof(haus<T>) * num_results);
        
        auto discard_pointer_st = static_cast<int32_t*>(discard_buffer.data());
        auto discard_pointer_l = static_cast<int64_t*>(discard_buffer.data());
        auto discard_pointer_t = static_cast<T*>(discard_buffer.data());

        auto out_zip = thrust::make_zip_iterator(
            thrust::make_tuple(
                discard_pointer_st,
                discard_pointer_st,
                discard_pointer_st,
                discard_pointer_st,
                discard_pointer_t,
                discard_pointer_t,
                out_real,
                discard_pointer_l
            )
        );

        auto out = make_haus_output_iterator(out_zip);

        thrust::inclusive_scan_by_key(
            rmm::exec_policy(stream)->on(stream),
            hausdorff_iter,
            hausdorff_iter + num_cartesian,
            hausdorff_iter,
            out,
            haus_key_compare<T>{},
            haus_reduce<T>{}
        );

        return result;
    }
};

} // namespace anonymous

namespace cuspatial {

std::unique_ptr<cudf::column>
directed_hausdorff_distance(cudf::column_view const& xs,
                            cudf::column_view const& ys,
                            cudf::column_view const& points_per_space,
                            rmm::mr::device_memory_resource *mr)
{
    CUSPATIAL_EXPECTS(xs.type() == ys.type(), "Inputs `xs` and `ys` must have same type.");
    CUSPATIAL_EXPECTS(xs.size() == ys.size(), "Inputs `xs` and `ys` must have same length.");

    CUSPATIAL_EXPECTS(not xs.has_nulls() and
                      not ys.has_nulls() and
                      not points_per_space.has_nulls(),
                      "Inputs must not have nulls.");

    CUSPATIAL_EXPECTS(xs.size() >= points_per_space.size(),
                      "At least one point is required for each space");

    hipStream_t stream = 0;

    return cudf::type_dispatcher(xs.type(), hausdorff_functor(),
                                 xs, ys, points_per_space, mr, stream);
}

} // namespace cuspatial
