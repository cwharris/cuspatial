#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required point_b_y applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "detail/cartesian_product_group_index_iterator.cuh"
#include "detail/hausdorff.cuh"

#include <utility/scatter_output_iterator.cuh>
#include <utility/size_from_offsets.cuh>

#include <cuspatial/error.hpp>

#include <cudf/column/column_device_view.cuh>
#include <cudf/column/column_factories.hpp>
#include <cudf/types.hpp>
#include <cudf/utilities/type_dispatcher.hpp>

#include <rmm/cuda_stream_view.hpp>
#include <rmm/device_uvector.hpp>
#include <rmm/exec_policy.hpp>

#include <thrust/binary_search.h>
#include <thrust/iterator/discard_iterator.h>
#include <thrust/iterator/transform_iterator.h>

#include <iostream>
#include <limits>
#include <memory>

namespace cuspatial {
namespace detail {
namespace {

struct partition_size_agg {
  uint32_t value;
  uint32_t group;

  inline constexpr partition_size_agg operator+(partition_size_agg other)
  {
    auto next_value = value + other.value;
    auto next_group = group + other.group;

    if (next_value > (1ull << 31)) {
      next_value = other.value;
      next_group = next_group + 1;
    }

    return {next_value, next_group};
  }
};

struct get_group {
  inline constexpr uint32_t operator()(partition_size_agg agg) { return agg.group; }
};

struct get_value {
  inline constexpr uint32_t operator()(partition_size_agg agg) { return agg.value; }
};

thrust::host_vector<uint32_t> get_partition_sizes(uint32_t num_offsets,
                                                  uint32_t num_elements,
                                                  uint32_t const* offsets,
                                                  rmm::cuda_stream_view stream)
{
  auto num_pairs  = num_offsets * num_offsets;
  auto sizes_iter = make_size_from_offset_iterator(num_offsets, num_elements, offsets);
  auto areas      = thrust::make_transform_iterator(
    thrust::make_counting_iterator(0), [num_offsets, sizes_iter] __device__(int idx) {
      return sizes_iter[idx / num_offsets] * sizes_iter[idx % num_offsets];
    });

  auto aggs = rmm::device_uvector<partition_size_agg>(num_pairs, stream);

  thrust::transform(
    rmm::exec_policy(stream), areas, areas + num_pairs, aggs.begin(), [] __device__(uint32_t area) {
      return partition_size_agg{area, 0};
    });

  thrust::inclusive_scan(rmm::exec_policy(stream),  //
                         aggs.begin(),
                         aggs.end(),
                         aggs.begin());

  auto num_partitions = aggs.back_element(stream).group + 1;

  std::cout << "num partitions: " << num_partitions << std::endl;

  auto partition_sizes = rmm::device_uvector<uint32_t>(num_partitions, stream);

  auto keys   = thrust::make_transform_iterator(aggs.begin(), get_group{});
  auto values = thrust::make_transform_iterator(aggs.begin(), get_value{});

  thrust::reduce_by_key(
    rmm::exec_policy(stream),
    keys,
    keys + num_pairs,
    values,
    thrust::make_discard_iterator(),
    partition_sizes.begin(),
    [] __device__(uint32_t a, uint32_t b) { return a == b; },
    [] __device__(uint32_t a, uint32_t b) { return b; });

  auto partition_sizes_h = thrust::host_vector<uint32_t>(num_partitions);

  std::cout << "partition_sizes_h: " << std::endl;

  hipMemcpy(partition_sizes_h.data(),
             partition_sizes.data(),
             partition_sizes_h.size() * sizeof(uint32_t),
             hipMemcpyDeviceToHost);

  for (uint32_t i = 0; i < partition_sizes_h.size(); i++) {
    std::cout << " partition_sizes_h[" << i << "] = " << partition_sizes_h[i] << std::endl;
  }

  return partition_sizes_h;
}

template <typename T>
struct hausdorff_accumulator_factory {
  cudf::column_device_view const xs;
  cudf::column_device_view const ys;

  hausdorff_acc<T> inline __device__ operator()(cartesian_product_group_index const& idx)
  {
    auto const a_idx = idx.group_a.offset + idx.element_a_idx;
    auto const b_idx = idx.group_b.offset + idx.element_b_idx;

    auto const distance = hypot(xs.element<T>(b_idx) - xs.element<T>(a_idx),
                                ys.element<T>(b_idx) - ys.element<T>(a_idx));

    return hausdorff_acc<T>{b_idx, b_idx, distance, distance, 0};
  }
};

struct hausdorff_functor {
  template <typename T, typename... Args>
  std::enable_if_t<not std::is_floating_point<T>::value, std::unique_ptr<cudf::column>> operator()(
    Args&&...)
  {
    CUSPATIAL_FAIL("Non-floating point operation is not supported");
  }

  template <typename T>
  std::enable_if_t<std::is_floating_point<T>::value, std::unique_ptr<cudf::column>> operator()(
    cudf::column_view const& xs,
    cudf::column_view const& ys,
    cudf::column_view const& space_offsets,
    rmm::cuda_stream_view stream,
    rmm::mr::device_memory_resource* mr)
  {
    auto const num_points  = static_cast<uint32_t>(xs.size());
    auto const num_spaces  = static_cast<uint32_t>(space_offsets.size());
    auto const num_results = static_cast<uint64_t>(num_spaces) * static_cast<uint64_t>(num_spaces);

    CUSPATIAL_EXPECTS(
      num_results < static_cast<uint64_t>(std::numeric_limits<cudf::size_type>::max()),
      "Matrix of spaces must be less than 2^31");

    if (num_results == 0) {
      return cudf::make_empty_column(cudf::data_type{cudf::type_to_id<T>()});
    }

    // ===== Partition Inputs ======================================================================

    auto partition_sizes =
      get_partition_sizes(num_spaces, num_points, space_offsets.begin<uint32_t>(), stream);

    // ===== Make Hausdorff Accumulator ============================================================

    auto gcp_iter = make_cartesian_product_group_index_iterator(
      num_points, num_spaces, space_offsets.begin<uint32_t>());

    auto d_xs = cudf::column_device_view::create(xs);
    auto d_ys = cudf::column_device_view::create(ys);

    auto hausdorff_acc_iter =
      thrust::make_transform_iterator(gcp_iter, hausdorff_accumulator_factory<T>{*d_xs, *d_ys});

    // ===== Materialize ===========================================================================

    auto result = cudf::make_fixed_width_column(cudf::data_type{cudf::type_to_id<T>()},
                                                static_cast<cudf::size_type>(num_results),
                                                cudf::mask_state::UNALLOCATED,
                                                stream,
                                                mr);

    auto result_temp = rmm::device_uvector<hausdorff_acc<T>>(num_results, stream);

    auto scatter_map = thrust::make_transform_iterator(
      gcp_iter, [num_spaces] __device__(cartesian_product_group_index const& idx) {
        // the given output is only a "result" if it is the last output for a given pair-of-spaces
        bool const is_result = idx.element_a_idx + 1 == idx.group_a.size &&  //
                               idx.element_b_idx + 1 == idx.group_b.size;

        if (not is_result) { return static_cast<uint32_t>(-1); }

        // the destination for the result is determined per- pair-of-spaces
        return idx.group_b.idx * num_spaces + idx.group_a.idx;
      });

    auto scatter_out = make_scatter_output_iterator(result_temp.begin(), scatter_map);

    auto gpc_key_iter = thrust::make_transform_iterator(
      gcp_iter, [] __device__(cartesian_product_group_index const& idx) {
        return thrust::make_pair(idx.group_a.idx, idx.group_b.idx);
      });

    for (uint32_t i = 0; i < partition_sizes.size(); i++) {
      thrust::inclusive_scan_by_key(rmm::exec_policy(stream),
                                    gpc_key_iter,
                                    gpc_key_iter + partition_sizes[i],
                                    hausdorff_acc_iter,
                                    scatter_out,
                                    thrust::equal_to<thrust::pair<uint32_t, uint32_t>>());

      gpc_key_iter += partition_sizes[i];
      scatter_out += partition_sizes[i];
    }

    thrust::transform(rmm::exec_policy(stream),
                      result_temp.begin(),
                      result_temp.end(),
                      result->mutable_view().begin<T>(),
                      [] __device__(hausdorff_acc<T> const& a) { return static_cast<T>(a); });

    return result;
  }
};

}  // namespace
}  // namespace detail

std::unique_ptr<cudf::column> directed_hausdorff_distance(cudf::column_view const& xs,
                                                          cudf::column_view const& ys,
                                                          cudf::column_view const& space_offsets,
                                                          rmm::mr::device_memory_resource* mr)
{
  CUSPATIAL_EXPECTS(xs.type() == ys.type(), "Inputs `xs` and `ys` must have same type.");
  CUSPATIAL_EXPECTS(xs.size() == ys.size(), "Inputs `xs` and `ys` must have same length.");

  CUSPATIAL_EXPECTS(not xs.has_nulls() and not ys.has_nulls() and not space_offsets.has_nulls(),
                    "Inputs must not have nulls.");

  CUSPATIAL_EXPECTS(xs.size() >= space_offsets.size(),
                    "At least one point is required for each space");

  return cudf::type_dispatcher(
    xs.type(), detail::hausdorff_functor(), xs, ys, space_offsets, rmm::cuda_stream_default, mr);
}

}  // namespace cuspatial
