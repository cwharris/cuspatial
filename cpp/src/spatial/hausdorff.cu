#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required point_b_y applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cuspatial/detail/hausdorff.cuh>

#include <cudf/types.hpp>
#include <cudf/utilities/type_dispatcher.hpp>
#include <cudf/column/column.hpp>
#include <cudf/column/column_device_view.cuh>
#include <cudf/column/column_factories.hpp>
#include <cudf/column/column_view.hpp>
#include <cuspatial/error.hpp>
#include <cudf/utilities/error.hpp>
#include <rmm/mr/device/device_memory_resource.hpp>
#include <rmm/thrust_rmm_allocator.h>
#include <rmm/device_buffer.hpp>

#include <thrust/iterator/transform_output_iterator.h>
#include <thrust/iterator/discard_iterator.h>
#include <thrust/gather.h>

#include <limits>
#include <iterator>
#include <iomanip>
#include <memory>
#include <ostream>
#include <type_traits>

template <class OutputIterator>
class haus_output_iterator_proxy;

// namespace thrust {
// namespace detail {
//     template <class OutputIterator>
//     struct is_proxy_reference<haus_output_iterator_proxy<OutputIterator>>
//         : public thrust::detail::true_type {};
// }
// }

namespace cuspatial {
namespace detail{
namespace {

using size_type = int32_t;

template<typename T>
constexpr auto magnitude_squared(T a, T b) {
    return a * a + b * b;
}

template<typename T>
std::unique_ptr<cudf::column> make_column(
    size_type size,
    hipStream_t stream = 0,
    rmm::mr::device_memory_resource* mr = rmm::mr::get_default_resource()
)
{
    auto tid = cudf::type_to_id<T>();

    return cudf::make_fixed_width_column(
        cudf::data_type{ tid },
        size,
        cudf::mask_state::UNALLOCATED,
        stream,
        mr);
}

struct size_from_offsets_functor
{
    cudf::column_device_view offsets;
    size_type end;

    size_type __device__ operator()(size_type idx)
    {
        auto curr_offset = offsets.element<size_type>(idx);
        auto next_idx = idx + 1;
        auto next_offset = next_idx >= offsets.size()
            ? end
            : offsets.element<size_type>(next_idx);

        return next_offset - curr_offset;
    }
};

template <typename OutputIterator>
class haus_output_iterator_proxy
{
  public:
    __host__ __device__
    haus_output_iterator_proxy(const OutputIterator& out, const OutputIterator& begin) : out(out), begin(begin)
    {
    }

    __thrust_exec_check_disable__
    template <typename T>
    __host__ __device__
    haus_output_iterator_proxy operator=(const T& x)
    {
        if (haus_dst(x) >= 0) {
            *(begin + haus_dst(x)) = x;
        }

        // *out = x;

        return *this;
    }

  private:
    OutputIterator out;
    OutputIterator begin;
};

template<typename OutputIterator>
class haus_output_iterator;

template <typename OutputIterator>
struct haus_output_iterator_base
{
    typedef thrust::iterator_adaptor
    <
        haus_output_iterator<OutputIterator>
      , OutputIterator
      , thrust::use_default
      , thrust::use_default
      , thrust::use_default
      , haus_output_iterator_proxy<OutputIterator>
    > type;
};

template<typename OutputIterator>
class haus_output_iterator : public haus_output_iterator_base<OutputIterator>::type
{
public:
    typedef typename
    haus_output_iterator_base<OutputIterator>::type
    super_t;

    friend class thrust::iterator_core_access;

    __host__ __device__
    haus_output_iterator(OutputIterator const& out, OutputIterator const& begin) : super_t(out), begin(begin) {}
    
 private:
    __host__ __device__
    typename super_t::reference dereference() const
    {
        return haus_output_iterator_proxy<OutputIterator>(this->base_reference(), begin);
    }

    OutputIterator begin;
};

template <typename OutputIterator>
haus_output_iterator<OutputIterator>
__host__ __device__
make_haus_output_iterator(OutputIterator out)
{
    return haus_output_iterator<OutputIterator>(out, out);
}

template<typename T, typename SpaceSizeIterator>
struct haus_travesal
{
    int64_t num_spaces;
    int64_t n;
    size_type const* o;
    size_type const* l;
    SpaceSizeIterator const s;
    cudf::column_device_view xs;
    cudf::column_device_view ys;

    haus<T> __device__ operator()(int64_t idx)
    {
        // ===== Reduction Key ===========
        int64_t haus_col = l[idx / n];
        int64_t ox = o[haus_col];
        int64_t sx = s[haus_col];
        int64_t ox_n = ox * n;

        int64_t haus_row = l[(idx - ox_n) / sx];
        int64_t oy = o[haus_row];
        int64_t sy = s[haus_row];

        // ===== Min/Max Key ==========
        int64_t haus_offset = ox_n + sx * oy;
        int64_t cell_idx = idx - haus_offset;
        int64_t cell_col = cell_idx / sy;

        // ===== Distance =============
        int64_t cell_offset = ox_n + oy + (n - sy) * cell_col + cell_idx;
        int64_t col = cell_offset / n;
        int64_t row = cell_offset % n;
        T a_x = xs.element<T>(row);
        T a_y = ys.element<T>(row);
        T b_x = xs.element<T>(col);
        T b_y = ys.element<T>(col);

        double distance_d = hypot(static_cast<double>(b_x - a_x),
                                  static_cast<double>(b_y - a_y));

        T distance = static_cast<T>(distance_d);

        int64_t elm = ox_n + (sx - 1) * n + oy + sy - 1;

        // ===== All ==================
        return haus<T>{
            thrust::make_tuple(haus_col, haus_row),
            elm == cell_offset
                ? haus_col * num_spaces + haus_row
                : -1,
            cell_col,
            cell_col,
            distance,
            distance,
            0,
            distance
        };
    }
};

struct hausdorff_functor
{
    template<typename T, typename... Args>
    std::enable_if_t<not std::is_floating_point<T>::value, std::unique_ptr<cudf::column>>
    operator()(Args&&...)
    {
        CUSPATIAL_FAIL("Non-floating point operation is not supported");
    }

    template<typename T>
    std::enable_if_t<std::is_floating_point<T>::value, std::unique_ptr<cudf::column>>
    operator()(cudf::column_view const& xs,
               cudf::column_view const& ys,
               cudf::column_view const& space_offsets,
               rmm::mr::device_memory_resource *mr,
               hipStream_t stream)
    {
        size_type num_points = xs.size();
        size_type num_spaces = space_offsets.size();
        int64_t num_results = static_cast<int64_t>(num_spaces) * static_cast<int64_t>(num_spaces);
        // int64_t num_results = static_cast<int64_t>(num_points) * static_cast<int64_t>(num_points);

        if (num_results == 0)
        {
            return make_column<T>(0, stream, mr);
        }

        // ===== Make Space Lookup ================================================================

        auto temp_space_lookup = rmm::device_vector<size_type>(num_points);

        thrust::scatter(
            rmm::exec_policy(stream)->on(stream),
            thrust::make_constant_iterator(1),
            thrust::make_constant_iterator(1) + num_spaces - 1,
            space_offsets.begin<size_type>() + 1,
            temp_space_lookup.begin()
        );

        thrust::inclusive_scan(
            rmm::exec_policy(stream)->on(stream),
            temp_space_lookup.cbegin(),
            temp_space_lookup.cend(),
            temp_space_lookup.begin()
        );

        // ===== Make Space Size Iterator =========================================================

        auto count = thrust::make_counting_iterator<int64_t>(0);

        auto d_space_offsets = cudf::column_device_view::create(space_offsets);

        auto space_offset_iterator = thrust::make_transform_iterator(
            count,
            size_from_offsets_functor { *d_space_offsets, xs.size() }
        );

        // ===== Make Cartesian Distances =========================================================

        auto d_xs = cudf::column_device_view::create(xs);
        auto d_ys = cudf::column_device_view::create(ys);

        auto num_cartesian = static_cast<int64_t>(num_points) * static_cast<int64_t>(num_points);

        auto hausdorff_iter = thrust::make_transform_iterator(
            count,
            haus_travesal<T, decltype(space_offset_iterator)>{
                num_spaces,
                num_points,
                space_offsets.data<size_type>(),
                temp_space_lookup.data().get(),
                space_offset_iterator,
                *d_xs,
                *d_ys
            }
        );

        // ===== Calculate ========================================================================

        std::unique_ptr<cudf::column> result = make_column<T>(num_results, stream, mr);

        auto out_real = result->mutable_view().begin<T>();

        auto discard_buffer = rmm::device_buffer(sizeof(haus<T>) * num_results);
        
        auto discard_pointer_st = static_cast<int32_t*>(discard_buffer.data());
        auto discard_pointer_l = static_cast<int64_t*>(discard_buffer.data());
        auto discard_pointer_t = static_cast<T*>(discard_buffer.data());
        auto discard_pointer_k = static_cast<thrust::tuple<int32_t, int32_t>*>(discard_buffer.data());

        auto out_zip = thrust::make_zip_iterator(
            thrust::make_tuple(
                discard_pointer_k,
                discard_pointer_l,
                discard_pointer_l,
                discard_pointer_l,
                discard_pointer_t,
                discard_pointer_t,
                discard_pointer_t,
                out_real
            )
        );

        auto out = make_haus_output_iterator(out_zip);

        thrust::inclusive_scan_by_key(
            rmm::exec_policy(stream)->on(stream),
            hausdorff_iter,
            hausdorff_iter + num_cartesian,
            hausdorff_iter,
            out,
            haus_key_compare<T>{},
            haus_reduce<T>{}
        );

        return result;
    }
};

} // namespace anonymous
} // namespace detail

std::unique_ptr<cudf::column>
directed_hausdorff_distance(cudf::column_view const& xs,
                            cudf::column_view const& ys,
                            cudf::column_view const& points_per_space,
                            rmm::mr::device_memory_resource *mr)
{
    CUSPATIAL_EXPECTS(xs.type() == ys.type(), "Inputs `xs` and `ys` must have same type.");
    CUSPATIAL_EXPECTS(xs.size() == ys.size(), "Inputs `xs` and `ys` must have same length.");

    CUSPATIAL_EXPECTS(not xs.has_nulls() and
                      not ys.has_nulls() and
                      not points_per_space.has_nulls(),
                      "Inputs must not have nulls.");

    CUSPATIAL_EXPECTS(xs.size() >= points_per_space.size(),
                      "At least one point is required for each space");

    hipStream_t stream = 0;

    return cudf::type_dispatcher(xs.type(), detail::hausdorff_functor(),
                                 xs, ys, points_per_space, mr, stream);
}

} // namespace cuspatial
