#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <memory>
#include <type_traits>
#include <thrust/device_vector.h>
#include <rmm/thrust_rmm_allocator.h>
#include <rmm/rmm.h>
#include <cudf/utilities/type_dispatcher.hpp>
#include <cudf/column/column.hpp>
#include <cudf/column/column_factories.hpp>
#include <cudf/column/column_view.hpp>
#include <utilities/legacy/cuda_utils.hpp>
#include <utility/utility.hpp>
#include <cuspatial/hausdorff.hpp>
#include <cuspatial/error.hpp>

namespace {

const unsigned int NUM_THREADS = 1024;

template <typename T>
__global__
void kernel_Hausdorff_Full(int num_traj,
                           T *xx,
                           T *yy,
                           uint32_t *pos,
                           T *results)
{
    int bidx = blockIdx.y * gridDim.x + blockIdx.x;

    if (bidx < num_traj*num_traj)
    {
        __shared__ T sdata[NUM_THREADS];

        int seg_id_left = bidx/num_traj;
        int seg_id_right =bidx%num_traj;

        sdata[threadIdx.x] = -1;

        __syncthreads();

        int start_left = seg_id_left == 0 ? 0 : pos[seg_id_left-1];
        int stop_left = pos[seg_id_left];

        int start_right = seg_id_right == 0 ? 0 : pos[seg_id_right-1];
        int stop_right = pos[seg_id_right];
        T dist = 1e20;
        int max_threads = 0;

        {
            max_threads = stop_left-start_left;
            if (threadIdx.x < max_threads)
            {
                T my_xx = xx[start_left+threadIdx.x];
                T my_yy = yy[start_left+threadIdx.x];
                for (int i = start_right; i < stop_right; i++)
                {
                    T other_xx = xx[i];
                    T other_yy = yy[i];
                    T new_dist = (my_xx-other_xx)*(my_xx-other_xx)
                        + (my_yy-other_yy)*(my_yy-other_yy);
                    dist= min(dist, new_dist);//dist < new_dist ? dist : new_dist;
                }
            }
        }

        if (dist > 1e10)
        {
            dist = -1;
        }

        if(threadIdx.x < max_threads)
        {
            sdata[threadIdx.x] = dist;
        }

        __syncthreads();

        //reduction
        for(int offset = blockDim.x / 2;
                offset > 0;
                offset >>= 1)
        {
            if(threadIdx.x < offset)
            {
                T tmp = sdata[threadIdx.x + offset];
                T tmp2 = sdata[threadIdx.x];
                sdata[threadIdx.x] = max(tmp2, tmp);
            }

            __syncthreads();
        }

        __syncthreads();

        if (threadIdx.x == 0)
        {
            results[bidx] = (sdata[0] >= 0) ? sqrt(sdata[0]) : 1e10;
        }
    }
}

struct hausdorff_functor
{
    template <typename T, typename... Args>
    std::enable_if_t<not std::is_floating_point<T>::value, std::unique_ptr<cudf::column>>
    operator()(Args...)

    {
        CUSPATIAL_FAIL("Non-floating point operation is not supported");
    }

    template <typename T>
    std::enable_if_t<std::is_floating_point<T>::value, std::unique_ptr<cudf::column>>
    const operator()(cudf::column_view const& x,
                     cudf::column_view const& y,
                     cudf::column_view const& vertex_counts,
                     rmm::mr::device_memory_resource *mr,
                     hipStream_t stream)
    {


        // rmm::exec_policy(stream)->on(stream)



        // // gdf_column d_matrix;
        // // memset(&d_matrix,0,sizeof(gdf_column));
        // // int num_set=vertex_counts.size;
        // // int block_sz = num_set*num_set;

        // // T *temp_matrix{nullptr};
        // // RMM_TRY( RMM_ALLOC(&temp_matrix, block_sz * sizeof(T), stream) );

        // // hipStream_t stream{0};
        // // auto exec_policy = rmm::exec_policy(stream);    

        // uint32_t *vertex_positions{nullptr};
        // RMM_TRY( RMM_ALLOC((void**)&vertex_positions, sizeof(uint32_t)*num_set, stream) );
        // uint32_t *vertex_counts_ptr=static_cast<uint32_t*>(vertex_counts.data);


        auto vertex_offsets = rmm::device_buffer(sizeof(uint32_t) * vertex_counts.size(), stream);
        thrust::inclusive_scan(rmm::exec_policy(stream)->on(stream),
                               vertex_counts.begin<uint32_t>(),
                               vertex_counts.end<uint32_t>(),
                               vertex_offsets.data());


        auto matrix = cudf::make_fixed_width_column(x.type(),
                                                    x.size() * x.size(),
                                                    cudf::mask_state::UNALLOCATED,
                                                    stream,
                                                    mr);

        return matrix;


        // int block_x = block_sz, block_y = 1;
        // if (block_sz > 65535)
        // {
        //     block_y = ceil((float)block_sz/65535.0);
        //     block_x = 65535;
        // }
        // dim3 grid(block_x, block_y);
        // dim3 block(NUM_THREADS);

        // kernel_Hausdorff_Full<T> <<< grid,block >>> (num_set,
        //     static_cast<T*>(x.data), static_cast<T*>(y.data),
        //     vertex_positions,temp_matrix);

        // CUDA_TRY( hipDeviceSynchronize() );
        // RMM_TRY( RMM_FREE(vertex_positions, stream) );

        // gdf_column_view_augmented(&d_matrix, temp_matrix, nullptr, block_sz,
        //                     x.dtype, 0,
        //                     gdf_dtype_extra_info{TIME_UNIT_NONE}, "hausdorff_matrix");

        // return d_matrix;
    }
};

} // namespace anonymous

/**
* @brief compute Hausdorff distances among all pairs of a set of trajectories
* see hausdorff.hpp
*/

namespace cuspatial {

std::unique_ptr<cudf::column>
directed_hausdorff_distance(cudf::column_view const& x,
                            cudf::column_view const& y,
                            cudf::column_view const& vertex_counts,
                            rmm::mr::device_memory_resource *mr)
{
    CUSPATIAL_EXPECTS(x.size() == y.size() ,
                      "`x` and `y` inputs must have same size");

    CUSPATIAL_EXPECTS(not x.has_nulls() &&
                      not y.has_nulls() &&
                      not vertex_counts.has_nulls(),
    	              "inputs must not be null");

    CUSPATIAL_EXPECTS(x.size() >= vertex_counts.size(),
                      "each trajectory must have at least one unique vertex");

    hipStream_t stream = 0;

    return cudf::experimental::type_dispatcher(x.type(), hausdorff_functor(),
                                               x, y, vertex_counts, mr, stream);
}

} // namespace cuspatial
